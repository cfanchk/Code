#include "hip/hip_runtime.h"
#include "../common/book.h"
#include "lock.h"

#define imin(a, b) (a<b?a:b)
#define sum_squares(x) (x*(x+1)*(2*x+1)/6)

const int N = 33 * 1024 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1)/threadsPerBlock);

__global__ void dot(Lock lock, float* a, float* b, float* c)
{
	__shared__ float cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int cacheIndex = threadIdx.x;
	float temp = 0;

	while(tid<N)
	{
		temp += a[tid] * b[tid];
		tid += blockDim.x * gridDim.x;
	}
	cache[cacheIndex] = temp;
	__syncthreads();

	int i = blockDim.x/2;
	while(i!=0)
	{
		if(cacheIndex<i)
			cache[cacheIndex] += cache[cacheIndex + i];
		__syncthreads();
		i /= 2;
	}

	if(cacheIndex == 0)
	{
		lock.lock();
		*c += cache[0];
		lock.unlock();
	}
}

int main()
{
	float* a, *b, c = 0;
	float* dev_a, *dev_b, *dev_c;

	a = (float*)malloc(N*sizeof(float));
	b = (float*)malloc(N*sizeof(float));

	HANDLE_ERROR(hipMalloc((void**)&dev_a, N*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, N*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c, sizeof(float)));

	for(int i=0; i<N; i++)
	{
		a[i] = i;
		b[i] = i * 2;
	}

	HANDLE_ERROR(hipMemcpy(dev_a, a, N*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_c, &c, sizeof(float), hipMemcpyHostToDevice));

	Lock lock;
	dot<<<blocksPerGrid,threadsPerBlock>>>(lock, dev_a, dev_b, dev_c);

	HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(float), hipMemcpyDeviceToHost));

	printf("GPU value: %.6g\nCorrect value: %.6g\n", c, 2*sum_squares((float)(N-1)));

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	free(a);
	free(b);

	return 0;
}
