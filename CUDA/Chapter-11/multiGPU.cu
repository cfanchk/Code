#include "hip/hip_runtime.h"
#include "../common/book.h"

#define imin(a,b) (a<b?a:b)

const int N = 33*1024*1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1) / threadsPerBlock);

struct DataStruct
{
	int deviceID;
	int size;
	float* a;
	float* b;
	float returnValue;
};

__global__ void dot(int size, float* a, float* b, float* c)
{
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < size)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x/2;
    while (i != 0) 
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

void* routine(void* pvoidData)
{
	DataStruct* data = (DataStruct*)pvoidData;
	HANDLE_ERROR(hipSetDevice(data->deviceID));

	int size = data->size;
	float* a, *b, c, *partial_c;
	float* dev_a, *dev_b, *dev_partial_c;

	a = data->a;
	b = data->b;
	partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

	HANDLE_ERROR(hipMalloc((void**)&dev_a, size*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, size*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, blocksPerGrid*sizeof(float)));

	HANDLE_ERROR(hipMemcpy(dev_a, a, size*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, size*sizeof(float), hipMemcpyHostToDevice));

	dot<<<blocksPerGrid,threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost));

	c = 0;
	for(int i=0; i<blocksPerGrid; i++)
		c += partial_c[i];

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_partial_c));

	free(partial_c);

	data->returnValue = 0;
	return 0;
}

int main()
{
	int deviceCount;
	HANDLE_ERROR(hipGetDeviceCount(&deviceCount));
	if(deviceCount<2)
	{
		printf("Only found %d GPU(s)!\n", deviceCount);
		return 1;
	}

	float* a = (float*)malloc(sizeof(float)*N);
	HANDLE_NULL(a);
	float* b = (float*)malloc(sizeof(float)*N);
	HANDLE_NULL(b);

	for(int i=0; i<N; i++)
	{
		a[i] = i;
		b[i] = i*2;
	}

	DataStruct data[2];

	data[0].deviceID = 0;
	data[0].size = N/2;
	data[0].a = a;
	data[0].b = b;
	
	data[1].deviceID = 1;
	data[1].size = N/2;
	data[1].a = a + N/2;
	data[1].b = b + N/2;
	
	CUTThread thread = start_thread(routine, &(data[0]));
	routine(&(data[1]));

	end_thread(thread);

	free(a);
	free(b);

	printf("Value calculated: %f.\n", data[0].returnValue + data[1].returnValue);

	return 0;
}
