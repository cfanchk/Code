#include "hip/hip_runtime.h"
#include "../common/book.h"

#define imin(a,b) (a<b?a:b)

const int N = 33*1024*1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N+threadsPerBlock-1) / threadsPerBlock);
            
__global__ void dot(int size, float* a, float* b, float* c)
{
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    while (tid < size)
    {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    cache[cacheIndex] = temp;
    __syncthreads();

    int i = blockDim.x/2;
    while (i != 0) 
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

float malloc_test(int size)
{
	hipEvent_t start, stop;
	float* a, *b, c, *partial_c;
	float* dev_a, *dev_b, *dev_partial_c;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	a = (float*)malloc(size*sizeof(float));
	b = (float*)malloc(size*sizeof(float));
	partial_c = (float*)malloc(blocksPerGrid*sizeof(float));

	HANDLE_ERROR(hipMalloc((void**)&dev_a, size*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b, size*sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&dev_partial_c, blocksPerGrid*sizeof(float)));

	for(int i=0; i<size; i++)
	{
		a[i] = i;
		b[i] = i*2;
	}

	HANDLE_ERROR(hipEventRecord(start, 0));
	HANDLE_ERROR(hipMemcpy(dev_a, a, size*sizeof(float), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, size*sizeof(float), hipMemcpyHostToDevice));

	dot<<<blocksPerGrid,threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);

	HANDLE_ERROR(hipMemcpy(partial_c, dev_partial_c, blocksPerGrid*sizeof(float), hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	c = 0;
	for(int i=0; i<blocksPerGrid; i++)
		c += partial_c[i];

	HANDLE_ERROR(hipFree(dev_a));
	HANDLE_ERROR(hipFree(dev_b));
	HANDLE_ERROR(hipFree(dev_partial_c));

	free(a);
	free(b);
	free(partial_c);

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	printf("Value calculated: %f\n", c);

	return elapsedTime;
}

float cuda_host_malloc_test(int size)
{
	hipEvent_t start, stop;
	float* a, *b, c, *partial_c;
	float* dev_a, *dev_b, *dev_partial_c;

	HANDLE_ERROR(hipEventCreate(&start));
	HANDLE_ERROR(hipEventCreate(&stop));

	HANDLE_ERROR(hipHostAlloc((void**)&a, size*sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&b, size*sizeof(float), hipHostMallocWriteCombined | hipHostMallocMapped));
	HANDLE_ERROR(hipHostAlloc((void**)&partial_c, size*sizeof(float), hipHostMallocMapped));

	for(int i=0; i<size; i++)
	{
		a[i] = i;
		b[i] = i*2;
	}

	HANDLE_ERROR(hipHostGetDevicePointer(&dev_a, a, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_b, b, 0));
	HANDLE_ERROR(hipHostGetDevicePointer(&dev_partial_c, partial_c, 0));

	HANDLE_ERROR(hipEventRecord(start, 0));

	dot<<<blocksPerGrid,threadsPerBlock>>>(size, dev_a, dev_b, dev_partial_c);
	HANDLE_ERROR(hipDeviceSynchronize());

	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));

	float elapsedTime;
	HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

	c = 0;
	for(int i=0; i<blocksPerGrid; i++)
		c += partial_c[i];

	HANDLE_ERROR(hipHostFree(a));
	HANDLE_ERROR(hipHostFree(b));
	HANDLE_ERROR(hipHostFree(partial_c));

	HANDLE_ERROR(hipEventDestroy(start));
	HANDLE_ERROR(hipEventDestroy(stop));

	printf("Value calculated: %f\n", c);

	return elapsedTime;
}

int main()
{
	hipDeviceProp_t prop;
	int whichDevice;
	HANDLE_ERROR(hipGetDevice(&whichDevice));
	HANDLE_ERROR(hipGetDeviceProperties(&prop, whichDevice));
	if(prop.canMapHostMemory != 1)
	{
		printf("Device cannot map memory!\n");
		return 1;
	}

	HANDLE_ERROR(hipSetDeviceFlags(hipDeviceMapHost));

	float elapsedTime = malloc_test(N);
	printf("Time using hipMalloc: %3.3f ms.\n", elapsedTime);

	elapsedTime = cuda_host_malloc_test(N);
	printf("Time using hipHostAlloc: %3.3f ms.\n", elapsedTime);
}
